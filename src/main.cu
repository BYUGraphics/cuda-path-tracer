#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

__global__ void testCUDA(int* _in, int _incrementBy){
	_in[threadIdx.x] += _incrementBy;
}

void printVec4(glm::vec4 _in){
	printf("(%f, %f, %f, %f)\n", _in.x, _in.y, _in.z, _in.w);
}

int main(int argc, char *argv[]){
	int i;
	
	printf("Hello, CUDA world!\n");
	
	if(argc < 2){
		printf("ERROR: please specify a scene to render: %s <scene.txt> [<output.png>]\n", argv[0]);
	}
	//get the path to the scene to render
	//get the name of the output image
	
	//load the scene
	//TODO
	
	//render the image
	//TODO
	
	//save the image to disk
	//TODO
	
	//some testing to make sure glm works
	glm::vec4 Position = glm::vec4(glm::vec3(0.0), 1.0);
	printVec4(Position);
	glm::mat4 Model = glm::translate(glm::mat4(1.0f), glm::vec3(1.0f));
	glm::vec4 Transformed = Model * Position;
	printVec4(Transformed);
	
	//some testing to make sure CUDA works
	int numBlocks = 1;
	int num = 5;
	int blockSize = num;
	int incrementBy = 2;
	int *h_a, *h_b;	//pointers to host memory
	int *d_a;		//pointer to device memory
	h_a = (int*)malloc(num * sizeof(int));	//allocate the first array on the host
	h_b = (int*)malloc(num * sizeof(int));	//allocate the second array on the host
	hipMalloc((void **) &d_a, num * sizeof(int));	//allocate memory on the device
	//initialize the host array a
	printf("Original array:\n");
	for(i = 0; i < num; i++){
		h_a[i] = i;
		printf("%d\n", h_a[i]);
	}
	//copy host array a to the device
	hipMemcpy(d_a, h_a, sizeof(int) * num, hipMemcpyHostToDevice);
	//run the CUDA kernel
	printf("Incrementing the array by %d\n", incrementBy);
	testCUDA<<<numBlocks, blockSize>>>(d_a, incrementBy);
	//copy the device memory back to the host
	hipMemcpy(h_b, d_a, sizeof(int) * num, hipMemcpyDeviceToHost);
	printf("New array:\n");
	for(i = 0; i < num; i++){
		printf("%d\n", h_b[i]);
	}
	
	
	return 0;
}


